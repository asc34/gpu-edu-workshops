#include "hip/hip_runtime.h"
/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "headers.h"

#define INDX(row,col,ld) (((col) * (ld)) + (row))

void costFunction( floatType_t *X, 
                   int const XRows, 
                   int const XCols,
                   floatType_t const *theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t const *theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   floatType_t const *Y, 
                   floatType_t *cost )
{

  floatType_t *tempMatrix, *a3;

printf("Xrows %d Xcols %d\n",XRows,XCols);
printf("t1row %d t1col %d\n",theta1Rows,theta1Cols);
printf("t2row %d t2col %d\n",theta2Rows,theta2Cols);

  for( int i = 0; i < XRows; i++ ) X[i] = (floatType_t) 1.0;
  
  tempMatrix = (floatType_t *) malloc( sizeof(floatType_t) *
                               ( XRows * (theta1Rows+1) + 
                                 XRows * (theta2Rows+1) ) );

  if( sizeof( floatType_t ) == 4 ) 
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 XRows, theta1Rows, theta1Cols,
                 1.0f, (float *) X, XRows,
                 (float *) theta1, theta1Rows, 0.0f,
                 (float *) &tempMatrix[INDX(0,1,XRows)], XRows );
  } /* end if */
  else
  {
  } /* end else */  

  for( int j = 1; j < theta1Rows+1; j++ )
    for( int i = 0; i < XRows; i++ )
      tempMatrix[INDX(i,j,XRows)] = 
        SIGMOID( tempMatrix[INDX(i,j,XRows)] );

  for( int i = 0; i < XRows; i++ ) 
    tempMatrix[INDX(i,0,XRows)] = (floatType_t) 1.0;

  a3 = &tempMatrix[INDX(0,theta2Cols+1,XRows)];

  if( sizeof( floatType_t ) == 4 )
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 XRows, theta2Rows, theta2Cols,
                 1.0f, (float *) tempMatrix, XRows,
                 (float *) theta2, theta2Rows, 0.0f,
                 (float *) a3, XRows );
//                 (float *) &tempMatrix[INDX(0,theta2Cols+1,XRows)], XRows );
  } /* end if */
  else
  { 
  } /* end else */

//  for( int i = 0; i < theta2Rows; i++ )
 //   printf("col %d val %f\n",i,a3[INDX(4999,i,XRows)] );
//    printf("col %d val %f\n",i,tempMatrix[INDX(4999,theta2Cols+1+i,XRows)] );

  
  for( int j = 0; j < theta2Rows; j++ )
    for( int i = 0; i < XRows; i++ )
      a3[INDX(i,j,XRows)] = 
        SIGMOID( a3[INDX(i,j,XRows)] );

//  for( int i = 0; i < theta2Rows; i++ )
 //   printf("col %d val %e\n",i,a3[INDX(4999,i,XRows)] );

  floatType_t yTemp[11];
  floatType_t jTemp = 0.0;

  for( int row = 0; row < XRows; row++ )
  {
    memset( yTemp, 0, sizeof(floatType_t) * 11 ); 
//   printf("row %d Y %f %d\n",row,Y[row],(int) Y[row] );
    yTemp[  (int)Y[row]  ] = (floatType_t) 1.0;
    for( int j = 1; j <= theta2Rows; j++ )
    {
//      printf("j %d val %f\n",j,-log(a3[INDX(row,j-1,XRows)])*yTemp[j]);
      jTemp += -log( a3[INDX(row,j-1,XRows)] ) * yTemp[j] 
             - ( log( (floatType_t) 1.0 - a3[INDX(row,j-1,XRows)] ) * 
                 ( (floatType_t) 1.0 - yTemp[j] ) ) ;
    } /* end for */
//    printf("row=%d jTemp is %f\n",row,jTemp);
  } /* end for */

  jTemp /= (floatType_t) XRows;
  printf("jTemp is %f %f\n",jTemp, jTemp / (floatType_t)XRows );
  free(tempMatrix);

} /* end costFunction */

void readMatrixFromFile( char *fileName, 
                         float *matrix, 
                         int const rows, 
                         int const cols )
{
  FILE *ifp;

  ifp = fopen( fileName, "r" );

  if( ifp == NULL ) 
  {
    fprintf(stderr, "Error opening file %s\n", fileName);
    exit(911);
  } /* end if */

  for( int row = 0; row < rows; row++ )
  {
    for( int col = 0; col < cols; col++ )
    {
      if( !fscanf( ifp, "%f", 
          &matrix[ INDX( row, col, rows ) ] ) )
      {
        fprintf(stderr,"error reading training matrix file \n");
        exit(911);
      } /* end if */
    } /* end for col */
  } /* end for row */

  fclose(ifp);
  return;
} /* end readMatrixFromFile */
