#include "hip/hip_runtime.h"
/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "headers.h"

void costFunction( floatType_t *X, 
                   int const XRows, 
                   int const XCols,
                   floatType_t const *theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t const *theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   floatType_t const *Y, 
                   floatType_t *cost )
{

  floatType_t *tempMatrix, *z2, *a2, *a3;

printf("Xrows %d Xcols %d\n",XRows,XCols);
printf("t1row %d t1col %d\n",theta1Rows,theta1Cols);
printf("t2row %d t2col %d\n",theta2Rows,theta2Cols);

  for( int i = 0; i < XRows; i++ ) X[i] = (floatType_t) 1.0;
  
  tempMatrix = (floatType_t *) malloc( sizeof(floatType_t) *
                               ( XRows * (theta1Rows+1) + 
                                 XRows * (theta1Rows+1) +
                                 XRows * (theta2Rows+1) ) );

  z2 = tempMatrix;
  a2 = &z2[INDX(XRows,theta1Rows,XRows)];
  a3 = &a2[INDX(XRows,theta1Rows+1,XRows)];

  if( sizeof( floatType_t ) == 4 ) 
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 XRows, theta1Rows, theta1Cols,
                 1.0f, (float *) X, XRows,
                 (float *) theta1, theta1Rows, 0.0f,
                 (float *) &z2[INDX(0,1,XRows)], XRows );
//                 (float *) &tempMatrix[INDX(0,1,XRows)], XRows );
    for( int j = 1; j < theta1Rows+1; j++ )
      for( int i = 0; i < XRows; i++ )
        a2[INDX(i,j,XRows)] = 
          sigmoid_f( z2[INDX(i,j,XRows)] );
  } /* end if */
  else
  {
  } /* end else */  



  for( int i = 0; i < XRows; i++ ) 
    a2[INDX(i,0,XRows)] = (floatType_t) 1.0;

//  a3 = &tempMatrix[INDX(0,theta2Cols+1,XRows)];

  if( sizeof( floatType_t ) == 4 )
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 XRows, theta2Rows, theta2Cols,
                 1.0f, (float *) a2, XRows,
                 (float *) theta2, theta2Rows, 0.0f,
                 (float *) a3, XRows );
//                 (float *) &tempMatrix[INDX(0,theta2Cols+1,XRows)], XRows );
    for( int j = 0; j < theta2Rows; j++ )
      for( int i = 0; i < XRows; i++ )
        a3[INDX(i,j,XRows)] = 
          sigmoid_f( a3[INDX(i,j,XRows)] );
  } /* end if */
  else
  { 
  } /* end else */

//  for( int i = 0; i < theta2Rows; i++ )
 //   printf("col %d val %f\n",i,a3[INDX(4999,i,XRows)] );
//    printf("col %d val %f\n",i,tempMatrix[INDX(4999,theta2Cols+1+i,XRows)] );

  


//  for( int i = 0; i < theta2Rows; i++ )
 //   printf("col %d val %e\n",i,a3[INDX(4999,i,XRows)] );

  floatType_t yTemp[11];
  floatType_t jTemp = 0.0;

  for( int row = 0; row < XRows; row++ )
  {
    memset( yTemp, 0, sizeof(floatType_t) * 11 ); 
//   printf("row %d Y %f %d\n",row,Y[row],(int) Y[row] );
    yTemp[  (int)Y[row]  ] = (floatType_t) 1.0;
    for( int j = 1; j <= theta2Rows; j++ )
    {
//      printf("j %d val %f\n",j,-log(a3[INDX(row,j-1,XRows)])*yTemp[j]);
      jTemp += -log( a3[INDX(row,j-1,XRows)] ) * yTemp[j] 
             - ( log( (floatType_t) 1.0 - a3[INDX(row,j-1,XRows)] ) * 
                 ( (floatType_t) 1.0 - yTemp[j] ) ) ;
    } /* end for */
//    printf("row=%d jTemp is %f\n",row,jTemp);
  } /* end for */

  jTemp /= (floatType_t) XRows;
  printf("jTemp is %f %f\n",jTemp, jTemp / (floatType_t)XRows );

  floatType_t *tempY, *delta3;
  tempY = (floatType_t *)malloc( sizeof(floatType_t)*11);
  delta3 = tempY;

  floatType_t *delta2;
  delta2 = (floatType_t *)malloc( sizeof(floatType_t) * theta2Cols );

  for( int row = 0; row < XRows; row++ )
  { 
    memset( tempY, 0, sizeof( floatType_t) * 11 );
    tempY[ (int) Y[row] ] = (floatType_t) 1.0;
    for( int j = 0; j < 10; j++ ) 
    {
      tempY[j+1] = a3[INDX(row,j,XRows)] - tempY[j+1];
//      printf("j %d delta3 %e\n",j+1, tempY[j+1] );
    } /* end for j */

    if( sizeof( floatType_t ) == 4 )
    {
      cblas_sgemv( CblasColMajor, CblasTrans,
                 theta2Rows, theta2Cols,
                 1.0f, theta2, theta2Rows, 
                 &delta3[1], 1, 0.0f,
                 delta2, 1 );

      for( int j = 1; j <= theta1Rows; j++ )
      {
        delta2[j] *= sigmoidGradient_f( z2[INDX(row,j,XRows)] );
      } /* end for */
    } /* end if */
    else
    { 
    } /* end else */



//    for( int j = 0; j <= theta1Rows; j++ )
 //    printf("j %d val %e\n",j,delta2[j] );
  //  printf("\n");

  } /* end for row */



  free(tempMatrix);
  free(tempY);
  free(delta2);
} /* end costFunction */

void readMatrixFromFile( char *fileName, 
                         float *matrix, 
                         int const rows, 
                         int const cols )
{
  FILE *ifp;

  ifp = fopen( fileName, "r" );

  if( ifp == NULL ) 
  {
    fprintf(stderr, "Error opening file %s\n", fileName);
    exit(911);
  } /* end if */

  for( int row = 0; row < rows; row++ )
  {
    for( int col = 0; col < cols; col++ )
    {
      if( !fscanf( ifp, "%f", 
          &matrix[ INDX( row, col, rows ) ] ) )
      {
        fprintf(stderr,"error reading training matrix file \n");
        exit(911);
      } /* end if */
    } /* end for col */
  } /* end for row */

  fclose(ifp);
  return;
} /* end readMatrixFromFile */
