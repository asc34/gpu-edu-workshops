#include "hip/hip_runtime.h"
/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include "headers.h"

void trainNetwork( floatType_t       *X, 
                   int         const Xexamples, 
                   int         const Xfeatures,
                   floatType_t       *theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t       *theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   floatType_t const *Y )
{
  floatType_t lambda = 0.3;
  floatType_t cost;
  floatType_t *theta1Grad, *theta2Grad;

  theta1Grad = (floatType_t *) malloc( sizeof(floatType_t) * 
                                theta1Rows * theta1Cols );

  theta2Grad = (floatType_t *) malloc( sizeof(floatType_t) * 
                                theta2Rows * theta2Cols );

#if 1
/* stochastic gradient descent */
  int iter = 0;
  while(iter < 20 )
  {
  for( int i = 0; i < 500; i++ )
  {
    for( int j = i; j < 5000; j+=500 )
    {
   //   int j = (int) ((double(rand()) / (double(RAND_MAX) + 1.0))*5000);
      
      costFunction( &X[INDX(0,j,Xfeatures)], 1, Xfeatures,
                    theta1, theta1Rows, theta1Cols, 
                    theta2, theta2Rows, theta2Cols,
                    &Y[j],
                    &cost, theta1Grad, theta2Grad );

//      printf("iter %d j %d cost is %.3e val %f\n",iter,j,cost,Y[j]);

      for( int i = 0; i < theta1Rows*theta1Cols; i++ )
        theta1[i] -= lambda * theta1Grad[i];

      for( int i = 0; i < theta2Rows*theta2Cols; i++ )
        theta2[i] -= lambda * theta2Grad[i];
//      printf("j %d val %f\n",j,Y[j]);
    } 
  } /* end for i */
  iter++;
    printf("|");
    fflush(stdout);
    if( iter % 72 == 0 ) printf("\n");
  } /* end while */
#endif
#if 0
/* gradient descent algorithm */

  int iter = 0;

  while( iter < 400 )
  {

  costFunction( X, Xexamples, Xfeatures,
                theta1, theta1Rows, theta1Cols, 
                theta2, theta2Rows, theta2Cols,
                Y,
                &cost, theta1Grad, theta2Grad );

//  printf("iter %d cost is %.3e\n",iter,cost);

  for( int i = 0; i < theta1Rows*theta1Cols; i++ )
    theta1[i] -= lambda * theta1Grad[i];

  for( int i = 0; i < theta2Rows*theta2Cols; i++ )
    theta2[i] -= lambda * theta2Grad[i];

    iter++;

    printf("|");
    fflush(stdout);
    if( iter % 72 == 0 ) printf("\n");
  } /* end while */
#endif
  printf("\nfinal cost value %.3e\n",cost);

} /* end trainNetwork */

void costFunction( floatType_t       *X, 
                   int         const Xexamples, 
                   int         const Xfeatures,
                   floatType_t const *theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t const *theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   floatType_t const *Y, 
                   floatType_t       *cost,
                   floatType_t       *theta1Grad,
                   floatType_t       *theta2Grad )
{

  floatType_t *tempMatrix, *z2, *a2, *a3;
//  floatType_t *theta1Grad, *theta2Grad;

//  printf("Xrows %d Xcols %d\n",XRows,XCols);
 // printf("t1row %d t1col %d\n",theta1Rows,theta1Cols);
  //printf("t2row %d t2col %d\n",theta2Rows,theta2Cols);

  for( int i = 0; i < Xexamples; i++ ) 
    X[INDX(0,i,Xfeatures)] = (floatType_t) 1.0;
  
  tempMatrix = (floatType_t *) malloc( sizeof(floatType_t) *
                               ( Xexamples * (theta1Rows+1) + 
                                 Xexamples * (theta1Rows+1) +
                                 Xexamples * (theta2Rows+1) ) );

  z2 = tempMatrix;
  a2 = &z2[INDX(Xexamples,theta1Rows,Xexamples)];
  a3 = &a2[INDX(Xexamples,theta1Rows+1,Xexamples)];


  if( sizeof( floatType_t ) == 4 ) 
  {
    cblas_sgemm( CblasColMajor, CblasTrans, CblasTrans,
                 Xexamples, theta1Rows, theta1Cols,
                 1.0f, (float *) X, Xfeatures,
                 (float *) theta1, theta1Rows, 0.0f,
                 (float *) &z2[INDX(0,1,Xexamples)], Xexamples );
//                 (float *) &tempMatrix[INDX(0,1,XRows)], XRows );
    for( int j = 1; j < theta1Rows+1; j++ )
      for( int i = 0; i < Xexamples; i++ )
        a2[INDX(i,j,Xexamples)] = 
          sigmoid_f( z2[INDX(i,j,Xexamples)] );
  } /* end if */
  else
  {
  } /* end else */  

  for( int i = 0; i < Xexamples; i++ ) 
    a2[INDX(i,0,Xexamples)] = (floatType_t) 1.0;

//  a3 = &tempMatrix[INDX(0,theta2Cols+1,XRows)];

  if( sizeof( floatType_t ) == 4 )
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 Xexamples, theta2Rows, theta2Cols,
                 1.0f, (float *) a2, Xexamples,
                 (float *) theta2, theta2Rows, 0.0f,
                 (float *) a3, Xexamples );
//                 (float *) &tempMatrix[INDX(0,theta2Cols+1,XRows)], XRows );
    for( int j = 0; j < theta2Rows; j++ )
      for( int i = 0; i < Xexamples; i++ )
        a3[INDX(i,j,Xexamples)] = 
          sigmoid_f( a3[INDX(i,j,Xexamples)] );
  } /* end if */
  else
  { 
  } /* end else */

//  for( int i = 0; i < theta2Rows; i++ )
 //   printf("col %d val %f\n",i,a3[INDX(4999,i,XRows)] );
//    printf("col %d val %f\n",i,tempMatrix[INDX(4999,theta2Cols+1+i,XRows)] );

  


//  for( int i = 0; i < theta2Rows; i++ )
 //   printf("col %d val %e\n",i,a3[INDX(4999,i,XRows)] );

  floatType_t yTemp[11];
  floatType_t jTemp = 0.0;

  for( int row = 0; row < Xexamples; row++ )
  {
    memset( yTemp, 0, sizeof(floatType_t) * 11 ); 
//   printf("row %d Y %f %d\n",row,Y[row],(int) Y[row] );
    yTemp[  (int)Y[row]  ] = (floatType_t) 1.0;
    for( int j = 1; j <= theta2Rows; j++ )
    {
//      printf("j %d val %f\n",j,-log(a3[INDX(row,j-1,XRows)])*yTemp[j]);
      jTemp += -log( a3[INDX(row,j-1,Xexamples)] ) * yTemp[j] 
             - ( log( (floatType_t) 1.0 - a3[INDX(row,j-1,Xexamples)] ) * 
                 ( (floatType_t) 1.0 - yTemp[j] ) ) ;
    } /* end for */
//    printf("row=%d jTemp is %f\n",row,jTemp);
  } /* end for */

  jTemp /= (floatType_t) Xexamples;
//  printf("jTemp is %f %f\n",jTemp, jTemp / (floatType_t)XRows );

  *cost = jTemp;

  floatType_t *tempY, *delta3;
  tempY = (floatType_t *)malloc( sizeof(floatType_t)*11);
  delta3 = tempY;

  floatType_t *delta2;
  delta2 = (floatType_t *)malloc( sizeof(floatType_t) * theta2Cols );

//  theta1Grad = (floatType_t *) malloc( sizeof(floatType_t) * 
 //                               theta1Rows * theta1Cols );


  memset( theta1Grad, 0, sizeof(floatType_t) * theta1Rows * theta1Cols );

//  theta2Grad = (floatType_t *) malloc( sizeof(floatType_t) * 
 //                               theta2Rows * theta2Cols );

  memset( theta2Grad, 0, sizeof(floatType_t) * theta2Rows * theta2Cols );

  for( int row = 0; row < Xexamples; row++ )
  { 
    memset( tempY, 0, sizeof( floatType_t) * 11 );
    tempY[ (int) Y[row] ] = (floatType_t) 1.0;

    for( int j = 0; j < 10; j++ ) 
    {
      tempY[j+1] = a3[INDX(row,j,Xexamples)] - tempY[j+1];
    } /* end for j */

    if( sizeof( floatType_t ) == 4 )
    {
      cblas_sgemv( CblasColMajor, CblasTrans,
                 theta2Rows, theta2Cols,
                 1.0f, theta2, theta2Rows, 
                 &delta3[1], 1, 0.0f,
                 delta2, 1 );

      for( int j = 1; j <= theta1Rows; j++ )
      {
        delta2[j] *= sigmoidGradient_f( z2[INDX(row,j,Xexamples)] );
      } /* end for */
    } /* end if */
    else
    { 
    } /* end else */

    for( int j = 0; j < theta1Cols; j++ )
    {
      for( int i = 0; i < theta1Rows; i++ )
      {
        theta1Grad[INDX(i,j,theta1Rows)] += 
          ( delta2[i+1] * X[INDX(j,row,Xfeatures)] );
//        printf("i %d j %d val %f\n",i,j,theta1Grad[INDX(i,j,theta1Rows)]);
      } /* end for i */    
    } /* end for j */

    for( int j = 0; j < theta2Cols; j++ )
    {
      for( int i = 0; i < theta2Rows; i++ )
      {
        theta2Grad[INDX(i,j,theta2Rows)] +=
          ( delta3[i+1] * a2[INDX(row,j,Xexamples)] );
//        printf("i %d j %d val %e\n",i,j,theta2Grad[INDX(i,j,theta2Rows)]);
      } /* end for i */
    } /* end for j */

  } /* end for row */

  floatType_t recip = (floatType_t) 1.0 / (floatType_t) Xexamples;

  for( int j = 0; j < theta1Cols; j++ )
  {
    for( int i = 0; i < theta1Rows; i++ )
    {
      theta1Grad[INDX(i,j,theta1Rows)] *= recip;
//      printf("i %d j %d val %e\n",i,j,theta1Grad[INDX(i,j,theta1Rows)]);
    } /* end for i */    
//    printf("\n");
  } /* end for j */

  for( int j = 0; j < theta2Cols; j++ )
  {
    for( int i = 0; i < theta2Rows; i++ )
    {
      theta2Grad[INDX(i,j,theta2Rows)] *= recip;
//      printf("i %d j %d val %e\n",i,j,theta2Grad[INDX(i,j,theta2Rows)]);
    } /* end for i */
 //   printf("\n");
  } /* end for j */

} /* end costFunction */

void predict( floatType_t *X, 
                   int const Xexamples, 
                   int const Xfeatures,
                   floatType_t const *theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t const *theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   int               *predictVector)
{

  floatType_t *tempMatrix, *z2, *a2, *a3;
//  floatType_t *theta1Grad, *theta2Grad;
 
  printf("Xrows %d Xcols %d\n",Xexamples,Xfeatures);
  printf("t1row %d t1col %d\n",theta1Rows,theta1Cols);
  printf("t2row %d t2col %d\n",theta2Rows,theta2Cols);

//  for( int i = 0; i < XRows; i++ ) X[i] = (floatType_t) 1.0;
  for( int i = 0; i < Xexamples; i++ ) 
    X[INDX(0,i,Xfeatures)] = (floatType_t) 1.0;

  tempMatrix = (floatType_t *) malloc( sizeof(floatType_t) *
                               ( Xexamples * (theta1Rows+1) + 
                                 Xexamples * (theta1Rows+1) +
                                 Xexamples * (theta2Rows+1) ) );

  z2 = tempMatrix;
  a2 = &z2[INDX(Xexamples,theta1Rows,Xexamples)];
  a3 = &a2[INDX(Xexamples,theta1Rows+1,Xexamples)];

  if( sizeof( floatType_t ) == 4 ) 
  {
    cblas_sgemm( CblasColMajor, CblasTrans, CblasTrans,
                 Xexamples, theta1Rows, theta1Cols,
                 1.0f, (float *) X, Xfeatures,
                 (float *) theta1, theta1Rows, 0.0f,
                 (float *) &z2[INDX(0,1,Xexamples)], Xexamples );
//                 (float *) &tempMatrix[INDX(0,1,XRows)], XRows );
    for( int j = 1; j < theta1Rows+1; j++ )
      for( int i = 0; i < Xexamples; i++ )
        a2[INDX(i,j,Xexamples)] = 
          sigmoid_f( z2[INDX(i,j,Xexamples)] );
  } /* end if */
  else
  {
  } /* end else */  



  for( int i = 0; i < Xexamples; i++ ) 
    a2[INDX(i,0,Xexamples)] = (floatType_t) 1.0;

//  a3 = &tempMatrix[INDX(0,theta2Cols+1,XRows)];

  if( sizeof( floatType_t ) == 4 )
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 Xexamples, theta2Rows, theta2Cols,
                 1.0f, (float *) a2, Xexamples,
                 (float *) theta2, theta2Rows, 0.0f,
                 (float *) a3, Xexamples );
//                 (float *) &tempMatrix[INDX(0,theta2Cols+1,XRows)], XRows );
    for( int j = 0; j < theta2Rows; j++ )
      for( int i = 0; i < Xexamples; i++ )
        a3[INDX(i,j,Xexamples)] = 
          sigmoid_f( a3[INDX(i,j,Xexamples)] );
  } /* end if */
  else
  { 
  } /* end else */

  for( int row = 0; row < Xexamples; row++ )
  {
    floatType_t max = -99.0;
    int         idx = -10;
    for( int i = 0; i < 10; i++ )
    {
      if( a3[INDX(row,i,Xexamples)] > max )
      {
        max = a3[INDX(row,i,Xexamples)];
        idx = i+1;
      } /* end if */
    } /* end for i */
    predictVector[row] = idx;
  } /* end row */

 
} /* end predict */ 


void readMatrixFromFile( char *fileName, 
                         float *matrix, 
                         int const rows, 
                         int const cols,
                         int const ld )
{
  FILE *ifp;

  ifp = fopen( fileName, "r" );

  if( ifp == NULL ) 
  {
    fprintf(stderr, "Error opening file %s\n", fileName);
    exit(911);
  } /* end if */

  for( int col = 0; col < cols; col++ )
  {
    for( int row = 0; row < rows; row++ )
    {
      if( !fscanf( ifp, "%f", 
          &matrix[ INDX( row, col, ld ) ] ) )
      {
        fprintf(stderr,"error reading training matrix file \n");
        exit(911);
      } /* end if */
    } /* end for row */
  } /* end for col */

  fclose(ifp);
  return;
} /* end readMatrixFromFile */
