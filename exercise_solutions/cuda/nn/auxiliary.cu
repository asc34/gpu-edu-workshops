#include "hip/hip_runtime.h"
/*
 *  Copyright 2014 NVIDIA Corporation
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *      http://www.apache.org/licenses/LICENSE-2.0
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 */

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hipDNN.h>
#include <hipblas.h>
#include "headers.h"

hipdnnHandle_t cudnnHandle;
hipdnnTensorDescriptor_t srcTensorDesc, destTensorDesc;
hipdnnTensorDescriptor_t srcDiffTensorDesc, destDiffTensorDesc;
hipblasHandle_t cublasHandle;

__global__ void k_updateDelta2( floatType_t       *delta2,
                                floatType_t const *z2,
                                int         const Xexamples,
                                int         const size )
{
  int tidx = blockDim.x * blockIdx.x + threadIdx.x;
  int tidy = blockDim.y * blockIdx.y + threadIdx.y;

//  if( tidy < Xexamples && tidx < size )
 //   delta2[INDX(tidx,tidy,size)] *= z2[INDX(tidy,tidx,Xexamples)];

  if( tidy == 0 && tidx < size )
  {
   for( int row = 0; row < Xexamples; row++ )
   {
//     for( int j = 0; j < size; j++ )
 //    {
       int j = tidx;
       delta2[INDX(j,row,size)] *= z2[INDX(row,j,Xexamples)];
  //   } /* end for */
   } /* end for */
  }
    

} /* end k_updateDelta2 */


__global__ void k_sigmoidGradient_f( floatType_t  *array,
                                     int    const size )
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if( tid < size )
    array[tid] = sigmoidGradient_f( array[tid] );
} /* end sigmoidGradient */

__global__ void  setYVec( floatType_t       *delta3, 
                          floatType_t const *Y, 
                          floatType_t const *a3,
                          int         const  Xexamples )
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if( tid < Xexamples )
  {
    delta3[INDX((int)Y[tid],tid,11)] = (floatType_t) 1.0;
    for( int j = 0; j < 10; j++ )
    {
      delta3[INDX(j+1,tid,11)] = a3[INDX(tid,j,Xexamples)]
                               - delta3[INDX(j+1,tid,11)];
    } /* end for j */
  }
  return;
} /* end setYVec */

__global__ void setVals( int rows, floatType_t *array )
{
  for( int i = 0; i < rows; i++ )
    array[i] = (floatType_t)i;
} /* end setVAls */

__global__ void initOne( int size, floatType_t *array )
{
  int tid = blockDim.x * blockIdx.x + threadIdx.x;
  if( tid < size )
    array[tid] = (floatType_t) 1.0;
  return;
} /* end initOne */

__global__ void printKernel( int rows, int cols, floatType_t *array )
{
  for( int j = 0; j < cols; j++ )
  {
    for( int i = 0; i < rows; i++ )
    {
      printf("row %d col %d value %e\n",i,j,array[INDX(i,j,rows)] );
    } /* end for */
  } /* end for */
} /* end print Kernel */

void printHost( int rows, int cols, floatType_t *array )
{
  for( int j = 0; j < cols; j++ )
  {
    for( int i = 0; i < rows; i++ )
    {
      printf("row %d col %d value %e\n",i,j,array[INDX(i,j,rows)] );
    } /* end for */
  } /* end for */
} /* end print Kernel */

void trainNetwork( floatType_t       *X, 
                   int         const Xexamples, 
                   int         const Xfeatures,
                   floatType_t       *theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t       *theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   floatType_t const *Y, 
                   float       const learningRate,
                   int         const iterations,
                   int         const batchSize )
{
  floatType_t lambda = learningRate;
  floatType_t cost;
  floatType_t *theta1Grad, *theta2Grad, *tempMatrix;



  checkCUDNN( hipdnnCreate( &cudnnHandle ) );
  checkCUDNN( hipdnnCreateTensorDescriptor( &srcTensorDesc ) );
  checkCUDNN( hipdnnCreateTensorDescriptor( &destTensorDesc ) );
  checkCUDNN( hipdnnCreateTensorDescriptor( &srcDiffTensorDesc ) );
  checkCUDNN( hipdnnCreateTensorDescriptor( &destDiffTensorDesc ) );
  checkCUBLAS( hipblasCreate( &cublasHandle ) );

  theta1Grad = (floatType_t *) malloc( sizeof(floatType_t) * 
                                theta1Rows * theta1Cols );

  theta2Grad = (floatType_t *) malloc( sizeof(floatType_t) * 
                                theta2Rows * theta2Cols );

  tempMatrix = (floatType_t *) malloc( sizeof(floatType_t) *
                               ( Xexamples * (theta1Rows+1) + //z2 
                                 Xexamples * (theta1Rows+1) + //a2
                                 Xexamples * (theta2Rows+1) + //a3
                                 Xexamples * (theta1Rows+1) + //delta2
                                 Xexamples * 11 ) );            //delta3

  floatType_t *d_tempMatrix;
  CUDA_CALL( hipMalloc( &d_tempMatrix, sizeof(floatType_t) *
                               ( Xexamples * (theta1Rows+1) + //z2 
                                 Xexamples * (theta1Rows+1) + //a2
                                 Xexamples * (theta2Rows+1) + //a3
                                 Xexamples * (theta1Rows+1) + //delta2
                                 Xexamples * 11 ) ) );            //delta3

  for( int i = 0; i < Xexamples; i++ ) 
    X[INDX(0,i,Xfeatures)] = (floatType_t) 1.0;

  floatType_t *d_X;
  CUDA_CALL( hipMalloc( &d_X, sizeof(floatType_t)*Xexamples*(Xfeatures+1)));
  CUDA_CALL( hipMemcpy( d_X, X, 
                         sizeof(floatType_t)*Xexamples*(Xfeatures+1),
                         hipMemcpyHostToDevice ) );

  floatType_t *d_Y;
  CUDA_CALL( hipMalloc( &d_Y, sizeof(floatType_t)*Xexamples) )
  CUDA_CALL( hipMemcpy( d_Y, Y, 
                         sizeof(floatType_t)*Xexamples,
                         hipMemcpyHostToDevice ) );

  floatType_t *d_theta1;
  CUDA_CALL( hipMalloc( &d_theta1, 
          sizeof(floatType_t) * theta1Rows * theta1Cols ) );

  CUDA_CALL( hipMemcpy( d_theta1, theta1,
                         sizeof(floatType_t)*theta1Rows*theta1Cols,
                         hipMemcpyHostToDevice ) );

  floatType_t *d_theta2;
  CUDA_CALL( hipMalloc( &d_theta2, 
          sizeof(floatType_t) * theta2Rows * theta2Cols ) );

  CUDA_CALL( hipMemcpy( d_theta2, theta2,
                         sizeof(floatType_t)*theta2Rows*theta2Cols,
                         hipMemcpyHostToDevice ) );

  floatType_t *d_theta1Grad, *d_theta2Grad;
  CUDA_CALL( hipMalloc( &d_theta1Grad, 
                         sizeof(floatType_t)*theta1Rows*theta1Cols ) );

  CUDA_CALL( hipMalloc( &d_theta2Grad,
                         sizeof(floatType_t)*theta2Rows*theta2Cols ) );

#if 1
/* stochastic gradient descent */
  int iter = 0;
//  int batchSize = 64;

//  printf("Learning rate Lambda is %f\n",lambda);
 // printf("Batchsize is %d\n",batchSize);

  while(iter < iterations )
  {
//  for( int i = 0; i < 500; i++ )
 // {
    for( int j = 0; j < Xexamples; j+=batchSize )
    {
   //   int j = (int) ((double(rand()) / (double(RAND_MAX) + 1.0))*5000);
#if 0
  CUDA_CALL( hipMemcpy( d_theta1, theta1,
                         sizeof(floatType_t)*theta1Rows*theta1Cols,
                         hipMemcpyHostToDevice ) );
#endif

//  CUDA_CALL( hipMemcpy( d_theta2, theta2,
 //                        sizeof(floatType_t)*theta2Rows*theta2Cols,
  //                       hipMemcpyHostToDevice ) );
      
      int tempBatchSize = min( batchSize, Xexamples - j );
 //     printf("before j %d tempBatchSize is %d\n",j,tempBatchSize);
//      costFunction( &d_X[INDX(0,j,Xfeatures)], batchSize, Xfeatures,
      costFunction( &d_X[INDX(0,j,Xfeatures)], tempBatchSize, Xfeatures,
                    d_theta1, theta1Rows, theta1Cols, 
                    d_theta2, theta2Rows, theta2Cols,
                    &d_Y[j],
                    &cost, d_theta1Grad, d_theta2Grad, 
                    d_tempMatrix );
//      printf("after j %d tempBatchsize is %d\n",j,tempBatchSize);
#if 0
  CUDA_CALL( hipMemcpy( theta1Grad, d_theta1Grad,
                         sizeof(floatType_t)*theta1Rows*theta1Cols,
                         hipMemcpyDeviceToHost ) );
#endif

 // CUDA_CALL( hipMemcpy( theta2Grad, d_theta2Grad,
  //                       sizeof(floatType_t)*theta2Rows*theta2Cols,
   //                      hipMemcpyDeviceToHost ) );

//      printf("iter %d j %d cost is %.3e val %f\n",iter,j,cost,Y[j]);
  floatType_t alpha = -lambda;
  checkCUBLAS( hipblasSaxpy( cublasHandle,
                            theta1Rows*theta1Cols,
                            &alpha,
                            d_theta1Grad, 1,
                            d_theta1, 1 ) ); 
#if 0
//      for( int i = 0; i < theta1Rows*theta1Cols; i++ )
 //       theta1[i] -= lambda * theta1Grad[i];
      cblas_saxpy( theta1Rows*theta1Cols, alpha, theta1Grad, 1, theta1, 1 );
#endif

  checkCUBLAS( hipblasSaxpy( cublasHandle,
                            theta2Rows*theta2Cols,
                            &alpha,
                            d_theta2Grad, 1,
                            d_theta2, 1 ) ); 
#if 0
//      for( int i = 0; i < theta2Rows*theta2Cols; i++ )
 //       theta2[i] -= lambda * theta2Grad[i];
      cblas_saxpy( theta2Rows*theta2Cols, alpha, theta2Grad, 1, theta2, 1 );
#endif
//      printf("j %d val %f\n",j,Y[j]);
//     exit(911);
    } 
 // } /* end for i */
  iter++;
    printf("|");
    fflush(stdout);
    if( iter % 72 == 0 ) printf("\n");
  } /* end while */
#endif
  CUDA_CALL( hipMemcpy( theta1, d_theta1,
                         sizeof(floatType_t)*theta1Rows*theta1Cols,
                         hipMemcpyDeviceToHost ) );
  CUDA_CALL( hipMemcpy( theta2, d_theta2,
                         sizeof(floatType_t)*theta2Rows*theta2Cols,
                         hipMemcpyDeviceToHost ) );
#if 0
/* gradient descent algorithm */

  int iter = 0;

  while( iter < 20 )
  {

  costFunction( X, Xexamples, Xfeatures,
                theta1, theta1Rows, theta1Cols, 
                theta2, theta2Rows, theta2Cols,
                Y,
                &cost, theta1Grad, theta2Grad );

  printf("iter %d cost is %.3e\n",iter,cost);

  for( int i = 0; i < theta1Rows*theta1Cols; i++ )
    theta1[i] -= lambda * theta1Grad[i];

  for( int i = 0; i < theta2Rows*theta2Cols; i++ )
    theta2[i] -= lambda * theta2Grad[i];

    iter++;

//    printf("|");
 //   fflush(stdout);
  //  if( iter % 72 == 0 ) printf("\n");
  } /* end while */
#endif
  printf("\nFinal cost value                      %.3e\n",cost);
  free(tempMatrix);
  free(theta1Grad);
  free(theta2Grad);
  CUDA_CALL( hipFree( d_tempMatrix ) );
  CUDA_CALL( hipFree( d_X ) );
  CUDA_CALL( hipFree( d_Y ) );
  CUDA_CALL( hipFree( d_theta1 ) );
  CUDA_CALL( hipFree( d_theta2 ) );
  CUDA_CALL( hipFree( d_theta1Grad ) );
  CUDA_CALL( hipFree( d_theta2Grad ) );

} /* end trainNetwork */

void costFunction( floatType_t       *d_X, 
                   int         const Xexamples, 
                   int         const Xfeatures,
                   floatType_t const *d_theta1, 
                   int         const theta1Rows,
                   int         const theta1Cols,
                   floatType_t const *d_theta2, 
                   int         const theta2Rows,
                   int         const theta2Cols,
                   floatType_t const *d_Y, 
                   floatType_t       *cost,
                   floatType_t       *d_theta1Grad,
                   floatType_t       *d_theta2Grad,
                   floatType_t       *d_tempMatrix )
{
#if 0
  floatType_t *z2, *a2, *a3;
  floatType_t *yTemp;
  floatType_t *delta2;
#endif
/* offset the pointers in the scratch memory */

//  z2 = tempMatrix;
//  a2 = &z2[INDX(Xexamples,theta1Rows,Xexamples)];
//  a3 = &a2[INDX(Xexamples,theta1Rows+1,Xexamples)];
//  yTemp = &a3[INDX(Xexamples,theta2Rows+1,Xexamples)];
//  delta2 = &yTemp[11];
#if 0
  z2 = tempMatrix;
  a2 = &z2[INDX(Xexamples,theta1Rows+1,Xexamples)];
  a3 = &a2[INDX(Xexamples,theta1Rows+1,Xexamples)];
  delta2 = &a3[INDX(Xexamples,theta2Rows+1,Xexamples)];
  yTemp = &delta2[INDX(Xexamples,theta1Rows+1,Xexamples)];
#endif
#if 0
  z2     = tempMatrix;
  a2     = &z2[Xexamples*(theta1Rows+1)];
  a3     = &a2[Xexamples*(theta1Rows+1)];
  delta2 = &a3[Xexamples*(theta2Rows+1)];
  yTemp  = &delta2[Xexamples*(theta1Rows+1)];
#endif

#if 0
  floatType_t *d_tempMatrix;
  CUDA_CALL( hipMalloc( &d_tempMatrix, sizeof(floatType_t) *
                               ( Xexamples * (theta1Rows+1) + //z2 
                                 Xexamples * (theta1Rows+1) + //a2
                                 Xexamples * (theta2Rows+1) + //a3
                                 Xexamples * (theta1Rows+1) + //delta2
                                 Xexamples * 11 ) ) );            //delta3
#endif


  floatType_t *d_z2, *d_a2, *d_a3, *d_yTemp, *d_delta2;

  d_z2     = d_tempMatrix;
  d_a2     = &d_z2[Xexamples*(theta1Rows+1)];
  d_a3     = &d_a2[Xexamples*(theta1Rows+1)];
  d_delta2 = &d_a3[Xexamples*(theta2Rows+1)];
  d_yTemp  = &d_delta2[Xexamples*(theta1Rows+1)];

#if 0
  d_z2 = d_tempMatrix;
  d_a2 = &d_z2[INDX(Xexamples,theta1Rows,Xexamples)];
  d_a3 = &d_a2[INDX(Xexamples,theta1Rows+1,Xexamples)];
  d_delta2 = &d_a3[INDX(Xexamples,theta2Rows+1,Xexamples)];
  d_yTemp = &d_delta2[INDX(Xexamples,theta1Rows+1,Xexamples)];
#endif

  checkCUDNN( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta1Rows+1,
                                         1,1) );

  checkCUDNN( hipdnnSetTensor4dDescriptor(destTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta1Rows+1,
                                         1,1) );

#if 0
  floatType_t *d_X;
  CUDA_CALL( hipMalloc( &d_X,
          sizeof(floatType_t)*Xexamples*Xfeatures ) );
#endif
//  floatType_t *d_theta1;
 // CUDA_CALL( hipMalloc( &d_theta1, 
  //        sizeof(floatType_t) * theta1Rows * theta1Cols ) );
#if 0
  floatType_t *d_theta2;
  CUDA_CALL( hipMalloc( &d_theta2, 
          sizeof(floatType_t) * theta2Rows * theta2Cols ) );
#endif
  float alpha = 1.0;
  float beta  = 0.0;

#if 1
  if( sizeof( floatType_t ) == 4 ) 
  {
//    cblas_sgemm( CblasColMajor, CblasTrans, CblasTrans,
 //                Xexamples, theta1Rows, theta1Cols,
  //               1.0f, (float *) X, Xfeatures,
   //              (float *) theta1, theta1Rows, 0.0f,
    //             (float *) &z2[INDX(0,1,Xexamples)], Xexamples );

//    printHost(100,1,&z2[INDX(0,1,Xexamples)] );

#if 0
    for( int i = Xexamples; i < Xexamples*(theta1Rows+1); i++ )
      a2[i] = sigmoid_f( z2[i] );
#endif
#if 1
#if 0
    CUDA_CALL( hipMemcpy( d_X, X,
                           sizeof(floatType_t)*Xexamples*Xfeatures,
                           hipMemcpyHostToDevice ) );
#endif
   // CUDA_CALL( hipMemcpy( d_theta1, theta1,
    //                       sizeof(floatType_t)*theta1Rows*theta1Cols,
     //                      hipMemcpyHostToDevice ) );
#if 0
    CUDA_CALL( hipMemcpy( d_theta2, theta2,
                           sizeof(floatType_t)*theta2Rows*theta2Cols,
                           hipMemcpyHostToDevice ) );
#endif
//    CUDA_CALL( hipMemcpy( d_srcData, z2, 
 //                          sizeof(floatType_t)*Xexamples*(theta1Rows+1),
  //                         hipMemcpyHostToDevice ) );
//    printf("rows %d cols %d\n",theta1Rows,theta1Cols);


//    printHost(Xfeatures*Xexamples,1,X);
 //   printKernel<<<1,1>>>( Xfeatures*Xexamples, 1, d_X );
  //  CUDA_CHECK()
   // CUDA_CALL( hipDeviceSynchronize() );

    checkCUBLAS( hipblasSgemm( cublasHandle, 
                              HIPBLAS_OP_T, HIPBLAS_OP_T,
                              Xexamples, theta1Rows, theta1Cols,
			      &alpha, d_X, Xfeatures,
                              d_theta1, theta1Rows, &beta,
                              &d_z2[INDX(0,1,Xexamples)], Xexamples ) );                              
//    CUDA_CALL( hipMemcpy( z2, d_z2, 
 //                          sizeof(floatType_t)*Xexamples*(theta1Rows+1),
  //                         hipMemcpyDeviceToHost ) );
    
//    setVals<<<1,1>>>(10,d_srcData );
 //   printKernel<<<1,1>>>( 5, 1, &d_srcData[5] );
  //  CUDA_CHECK()
   // CUDA_CALL( hipDeviceSynchronize() );
                                

    checkCUDNN( hipdnnActivationForward( cudnnHandle,
                                        HIPDNN_ACTIVATION_SIGMOID,
                                        &alpha,
                                        srcTensorDesc, d_z2,
                                        &beta,
                                        destTensorDesc, d_a2 ) );
#endif
//exit(911);

  } /* end if */
  else
  {
  } /* end else */  

#if 1
    initOne<<< Xexamples/256 + 1, 256 >>>( Xexamples, d_a2 );
    CUDA_CHECK()
    CUDA_CALL( hipDeviceSynchronize() );
//    CUDA_CALL( hipMemcpy( a2, d_a2, 
 //                          sizeof(floatType_t)*Xexamples*(theta1Rows+1),
  //                         hipMemcpyDeviceToHost ) );
#endif
#if 0
  for( int i = 0; i < Xexamples; i++ ) 
    a2[INDX(i,0,Xexamples)] = (floatType_t) 1.0;
#endif

  if( sizeof( floatType_t ) == 4 )
  {
#if 0
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 Xexamples, theta2Rows, theta2Cols,
                 1.0f, (float *) a2, Xexamples,
                 (float *) theta2, theta2Rows, 0.0f,
                 (float *) a3, Xexamples );
#endif
#if 1
    checkCUBLAS( hipblasSgemm( cublasHandle, 
                              HIPBLAS_OP_N, HIPBLAS_OP_T,
                              Xexamples, theta2Rows, theta2Cols,
			      &alpha, d_a2, Xexamples,
                              d_theta2, theta2Rows, &beta,
                              d_a3, Xexamples ) );                              

  checkCUDNN( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta2Rows,
                                         1,1) );

  checkCUDNN( hipdnnSetTensor4dDescriptor(destTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta2Rows,
                                         1,1) );

    checkCUDNN( hipdnnActivationForward( cudnnHandle,
                                        HIPDNN_ACTIVATION_SIGMOID,
                                        &alpha,
                                        srcTensorDesc, d_a3,
                                        &beta,
                                        destTensorDesc, d_a3 ) );
   // CUDA_CALL( hipMemcpy( a3, d_a3, 
    //                       sizeof(floatType_t)*theta2Rows*Xexamples,
     //                      hipMemcpyDeviceToHost ) );

#endif
#if 0
      for( int i = 0; i < theta2Rows*Xexamples; i++ )
        a3[i] = sigmoid_f( a3[i] );
#endif

  } /* end if */
  else
  { 
  } /* end else */

/* enable the following code if you wish to calculate the forward cost 
   not strictly necessary to generate the gradients
*/
#if 0
  floatType_t jTemp = 0.0;
  for( int row = 0; row < Xexamples; row++ )
  {
    memset( yTemp, 0, sizeof(floatType_t) * 11 ); 
    yTemp[  (int)Y[row]  ] = (floatType_t) 1.0;
    for( int j = 1; j <= theta2Rows; j++ )
    {
      jTemp += -log( a3[INDX(row,j-1,Xexamples)] ) * yTemp[j] 
             - ( log( (floatType_t) 1.0 - a3[INDX(row,j-1,Xexamples)] ) * 
                 ( (floatType_t) 1.0 - yTemp[j] ) ) ;
    } /* end for */
  } /* end for */

  jTemp /= (floatType_t) Xexamples;
  *cost = jTemp;
#endif
#endif

#if 1
//  floatType_t *delta3;
//  delta3 = yTemp;
//  memset( delta3, 0, sizeof( floatType_t) * 11 * Xexamples );

  floatType_t *d_delta3;
  d_delta3 = d_yTemp;
  CUDA_CALL( hipMemset( d_delta3, 0, sizeof(floatType_t)*11*Xexamples ) );

#if 0
  floatType_t *d_Y;
  CUDA_CALL( hipMalloc( &d_Y, sizeof(floatType_t) * Xexamples ) );

  CUDA_CALL( hipMemcpy( d_Y, Y, sizeof(floatType_t) * Xexamples,
                         hipMemcpyHostToDevice ) );
#endif
#if 0
  for( int row = 0; row < Xexamples; row++ )
  {
    delta3[INDX((int)Y[row],row,11)] = (floatType_t) 1.0;
  }
#endif
  setYVec<<< Xexamples/256+1, 256 >>>( d_delta3, d_Y, d_a3, Xexamples );
  CUDA_CHECK()
  CUDA_CALL( hipDeviceSynchronize() );

//  CUDA_CALL( hipMemcpy( delta3, d_delta3, sizeof(floatType_t)*11*Xexamples,
 //                        hipMemcpyDeviceToHost ) );
#if 0
  for( int row = 0; row < Xexamples; row++ )
  {
    for( int j = 0; j < 10; j++ )
    {
      delta3[INDX(j+1,row,11)] = a3[INDX(row,j,Xexamples)]
                               - delta3[INDX(j+1,row,11)];
    } /* end for j */
  } /* end for */
#endif
  if( sizeof( floatType_t ) == 4 )
  {

#if 1    
    checkCUBLAS( hipblasSgemm( cublasHandle, 
                              HIPBLAS_OP_T, HIPBLAS_OP_N,
                              theta2Cols, Xexamples, theta2Rows,
			      &alpha, (float *)d_theta2, theta2Rows,
                              (float *)&d_delta3[1], 11, &beta,
                              (float *)d_delta2, theta1Rows+1 ) );   

  //  CUDA_CALL( hipMemcpy( delta2, d_delta2, 
   //                        sizeof(floatType_t)*Xexamples*(theta1Rows+1),
    //                       hipMemcpyDeviceToHost ) );
#endif
#if 0
    cblas_sgemm( CblasColMajor, CblasTrans, CblasNoTrans,
                 theta2Cols, Xexamples, theta2Rows,
                 1.0f, theta2, theta2Rows,
                 &delta3[1],11, 0.0f,
                 delta2,theta1Rows+1);
#endif
#if 0
  checkCUDNN( hipdnnSetTensor4dDescriptor(srcTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta1Rows+1,
                                         1,1) );

  checkCUDNN( hipdnnSetTensor4dDescriptor(destTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta1Rows+1,
                                         1,1) );

  checkCUDNN( hipdnnSetTensor4dDescriptor(srcDiffTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta1Rows+1,
                                         1,1) );

  checkCUDNN( hipdnnSetTensor4dDescriptor(destDiffTensorDesc,
                                         HIPDNN_TENSOR_NCHW,
                                         HIPDNN_DATA_FLOAT,
                                         Xexamples,
                                         theta1Rows+1,
                                         1,1) );

  checkCUDNN( hipdnnActivationBackward( cudnnHandle, 
                                       HIPDNN_ACTIVATION_SIGMOID,
				       &alpha,
                                       srcTensorDesc, d_z2,
                                       srcDiffTensorDesc, d_delta2,
                                       destTensorDesc, d_z2, 
                                       &beta,
                                       destDiffTensorDesc, d_delta2 ) );

//   CUDA_CALL( hipMemcpy( delta2, d_delta2,
 //                         sizeof(floatType_t) * Xexamples*(theta1Rows+1),
  //                        hipMemcpyDeviceToHost ) );
#endif
#if 1
#if 0
   for( int i = 0; i < Xexamples*(theta1Rows+1); i++ )
     z2[i] = sigmoidGradient_f( z2[i] );
#endif
   dim3 threads(256,1,1);
   dim3 blocks(Xexamples*(theta1Rows+1)+1/threads.x,1,1);
   k_sigmoidGradient_f<<< blocks, threads >>>( d_z2, Xexamples*(theta1Rows+1) );
   CUDA_CHECK()
   CUDA_CALL( hipDeviceSynchronize() );
   
 //  CUDA_CALL( hipMemcpy( z2, d_z2, 
  //                        sizeof(floatType_t)*Xexamples*(theta1Rows+1),
   //                       hipMemcpyDeviceToHost ) );

   dim3 t1(256,256,1); 
   dim3 b1((theta1Rows+1)/t1.x + 1, Xexamples/t1.y + 1, 1 );

   k_updateDelta2<<< blocks,threads >>>( d_delta2, d_z2, Xexamples, theta1Rows+1 );
   CUDA_CHECK()
   CUDA_CALL( hipDeviceSynchronize() );

//   CUDA_CALL( hipMemcpy( delta2, d_delta2, 
 //                         sizeof(floatType_t)*Xexamples*(theta1Rows+1),
  //                        hipMemcpyDeviceToHost ) );

//    delta2[INDX(tidx,tidy,size)] *= z2[INDX(tidy,tidx,Xexamples)];
#if 0
   for( int row = 0; row < Xexamples; row++ )
   {
     for( int j = 0; j < theta1Rows+1; j++ )
     {
       delta2[INDX(j,row,theta1Rows+1)] *= z2[INDX(row,j,Xexamples)];
     } /* end for */
   } /* end for */
#endif
#endif

  } /* end if */
  else
  {
  } /* end else */

  floatType_t recip = (floatType_t) 1.0 / (floatType_t) Xexamples;
#if 0
  floatType_t *d_theta1Grad;
  CUDA_CALL( hipMalloc( &d_theta1Grad, 
                         sizeof(floatType_t)*theta1Rows*theta1Cols ) );
#endif

#if 0
  floatType_t *d_theta2Grad;
  CUDA_CALL( hipMalloc( &d_theta2Grad,
                         sizeof(floatType_t)*theta2Rows*theta2Cols ) );
#endif
    checkCUBLAS( hipblasSgemm( cublasHandle, 
                              HIPBLAS_OP_N, HIPBLAS_OP_T,
                              theta1Rows, theta1Cols, Xexamples,
			      &recip, (float *)&d_delta2[1], theta1Rows+1,
                              (float *)d_X, Xfeatures, 
                              &beta, (float *)d_theta1Grad, theta1Rows ) );   
#if 0
  CUDA_CALL( hipMemcpy( theta1Grad, d_theta1Grad, 
                         sizeof(floatType_t)*theta1Rows*theta1Cols,
                         hipMemcpyDeviceToHost ) );
#endif
#if 0
  cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
               theta1Rows, theta1Cols, Xexamples,
               recip, (float *) &delta2[1], theta1Rows+1,
               X, Xfeatures,
               0.0f, (float *) theta1Grad, theta1Rows );
#endif
    checkCUBLAS( hipblasSgemm( cublasHandle, 
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              theta2Rows, theta2Cols, Xexamples,
			      &recip, (float *)&d_delta3[1], 11,
                              (float *)d_a2, Xexamples, 
                              &beta, (float *)d_theta2Grad, theta2Rows ) );   
#if 0
  CUDA_CALL( hipMemcpy( theta2Grad, d_theta2Grad,
                         sizeof(floatType_t)*theta2Rows*theta2Cols,
                         hipMemcpyDeviceToHost ) );
#endif
#if 0
  cblas_sgemm( CblasColMajor, CblasNoTrans, CblasNoTrans,
               theta2Rows, theta2Cols, Xexamples,
               recip, (float *) &delta3[1], 11,
               (float *) a2, Xexamples, 0.0f,
               (float *) theta2Grad, theta2Rows );
#endif

#endif

#if 0
  floatType_t *delta3;
  delta3 = yTemp;


  memset( theta1Grad, 0, sizeof(floatType_t) * theta1Rows * theta1Cols );

  memset( theta2Grad, 0, sizeof(floatType_t) * theta2Rows * theta2Cols );

  for( int row = 0; row < Xexamples; row++ )
  { 
    memset( delta3, 0, sizeof( floatType_t) * 11 );
    delta3[ (int) Y[row] ] = (floatType_t) 1.0;
#if 1
    for( int j = 0; j < 10; j++ ) 
    {
      delta3[j+1] = a3[INDX(row,j,Xexamples)] - delta3[j+1];
    } /* end for j */

    if( sizeof( floatType_t ) == 4 )
    {
      cblas_sgemv( CblasColMajor, CblasTrans,
                 theta2Rows, theta2Cols,
                 1.0f, theta2, theta2Rows, 
                 &delta3[1], 1, 0.0f,
                 delta2, 1 );

      for( int j = 1; j <= theta1Rows; j++ )
      {
        delta2[j] *= sigmoidGradient_f( z2[INDX(row,j,Xexamples)] );
      } /* end for */
    } /* end if */
    else
    { 
    } /* end else */
#endif
#if 1
    for( int j = 0; j < theta1Cols; j++ )
    {
      for( int i = 0; i < theta1Rows; i++ )
      {
        theta1Grad[INDX(i,j,theta1Rows)] += 
          ( delta2[i+1] * X[INDX(j,row,Xfeatures)] );
      } /* end for i */    
    } /* end for j */

    for( int j = 0; j < theta2Cols; j++ )
    {
      for( int i = 0; i < theta2Rows; i++ )
      {
        theta2Grad[INDX(i,j,theta2Rows)] +=
          ( delta3[i+1] * a2[INDX(row,j,Xexamples)] );
      } /* end for i */
    } /* end for j */
#endif
  } /* end for row */

  floatType_t recip = (floatType_t) 1.0 / (floatType_t) Xexamples;

//  for( int j = 0; j < theta1Cols; j++ )
 // {
  //  for( int i = 0; i < theta1Rows; i++ )
   // {
    //  theta1Grad[INDX(i,j,theta1Rows)] *= recip;
  //  } /* end for i */    
//  } /* end for j */

  for( int i = 0; i < theta1Rows*theta1Cols; i++ )
    theta1Grad[i] *= recip;

//  for( int j = 0; j < theta2Cols; j++ )
 // {
  //  for( int i = 0; i < theta2Rows; i++ )
   // {
    //  theta2Grad[INDX(i,j,theta2Rows)] *= recip;
  //  } /* end for i */
//  } /* end for j */
  for( int i = 0; i < theta2Cols*theta2Rows; i++ )
    theta2Grad[i] *= recip;
#endif

//  CUDA_CALL( hipFree( d_tempMatrix ) );
//  CUDA_CALL( hipFree( d_X ) );
//  CUDA_CALL( hipFree( d_theta1 ) );
 // CUDA_CALL( hipFree( d_theta2 ) );
} /* end costFunction */

void predict(floatType_t       *X, 
             int         const Xexamples, 
             int         const Xfeatures,
             floatType_t const *theta1, 
             int         const theta1Rows,
             int         const theta1Cols,
             floatType_t const *theta2, 
             int         const theta2Rows,
             int         const theta2Cols,
             int               *predictVector)
{

  floatType_t *tempMatrix, *z2, *a2, *a3;
 
  for( int i = 0; i < Xexamples; i++ ) 
    X[INDX(0,i,Xfeatures)] = (floatType_t) 1.0;

  tempMatrix = (floatType_t *) malloc( sizeof(floatType_t) *
                               ( Xexamples * (theta1Rows+1) + 
                                 Xexamples * (theta1Rows+1) +
                                 Xexamples * (theta2Rows+1) ) );

  z2 = tempMatrix;
  a2 = &z2[INDX(Xexamples,theta1Rows,Xexamples)];
  a3 = &a2[INDX(Xexamples,theta1Rows+1,Xexamples)];

  if( sizeof( floatType_t ) == 4 ) 
  {
    cblas_sgemm( CblasColMajor, CblasTrans, CblasTrans,
                 Xexamples, theta1Rows, theta1Cols,
                 1.0f, (float *) X, Xfeatures,
                 (float *) theta1, theta1Rows, 0.0f,
                 (float *) &z2[INDX(0,1,Xexamples)], Xexamples );
    for( int j = 1; j < theta1Rows+1; j++ )
      for( int i = 0; i < Xexamples; i++ )
        a2[INDX(i,j,Xexamples)] = 
          sigmoid_f( z2[INDX(i,j,Xexamples)] );
  } /* end if */
  else
  {
  } /* end else */  



  for( int i = 0; i < Xexamples; i++ ) 
    a2[INDX(i,0,Xexamples)] = (floatType_t) 1.0;

  if( sizeof( floatType_t ) == 4 )
  {
    cblas_sgemm( CblasColMajor, CblasNoTrans, CblasTrans,
                 Xexamples, theta2Rows, theta2Cols,
                 1.0f, (float *) a2, Xexamples,
                 (float *) theta2, theta2Rows, 0.0f,
                 (float *) a3, Xexamples );
    for( int j = 0; j < theta2Rows; j++ )
      for( int i = 0; i < Xexamples; i++ )
        a3[INDX(i,j,Xexamples)] = 
          sigmoid_f( a3[INDX(i,j,Xexamples)] );
  } /* end if */
  else
  { 
  } /* end else */

  for( int row = 0; row < Xexamples; row++ )
  {
    floatType_t max = -99.0;
    int         idx = -10;
    for( int i = 0; i < 10; i++ )
    {
      if( a3[INDX(row,i,Xexamples)] > max )
      {
        max = a3[INDX(row,i,Xexamples)];
        idx = i+1;
      } /* end if */
    } /* end for i */
    predictVector[row] = idx;
  } /* end row */

 
} /* end predict */ 

void readCommandLineArgs( int    argc, 
                          char   *argv[],
                          float  *learningRate,
                          int    *batchSize,
                          int    *iterations,
                          int    *sizeHiddenLayer )
{
/* read command line input */
  switch( argc )
  {
    case 1:
      *learningRate = 0.3;
      *batchSize = 50;
      *iterations = 1;
      *sizeHiddenLayer = 25;
      break;
    case 2:
      if( strcmp( argv[1],"-h" ) == 0 )
      {
        printf("Usage: ./x.nn -h for this message\n");
        printf("Usage: ./x.nn <learningRate:float> <batchSize:int> <iterations:int> <hiddenLayerSize:int>\n");
        exit(911);
      } /* end for */
      break;
    case 5:
      *learningRate = atof( argv[1] );
      if( *learningRate == 0.0f )
      {
        printf("Invalid learning rate %s\n", argv[1] );
        *learningRate = 0.3;
        printf("Defaulting to %e\n", *learningRate );
      } /* end if */

      *batchSize = atoi( argv[2] );
      if( *batchSize <= 0 )
      {
        printf("Invalid batchSize %s\n", argv[2] );
        *batchSize = 50;
        printf("Defaulting to %d\n",*batchSize );
      } /* end if */

      *iterations = atoi( argv[3] );
      if( *iterations <= 0 )
      {
        printf("Invalid iteration size %s\n", argv[3] );
        *iterations = 1;
        printf("Defaulting to %d\n",*iterations);
      } /* end if */

      *sizeHiddenLayer = atoi( argv[4] );
      if( *sizeHiddenLayer <= 0 )
      {
        printf("Invalid hidden layer size %s\n", argv[4] );
        *sizeHiddenLayer = 25;
        printf("Defaulting to %d\n",*sizeHiddenLayer );
      } /* end if */
      break;
    default:
      printf("Undefined command-line args\n");
      printf("Usage: ./x.nn -h for this message\n");
      printf("Usage: ./x.nn <learningRate:float> <batchSize:int> <iterations:int> <hiddenLayerSize:int>\n");
      exit(911);
      break;

  } /* end switch */

/* print some initial stuff */
  printf("Learning rate lambda is               %.3e\n",*learningRate);
  printf("Batchsize is                          %d\n",*batchSize);
  printf("Number of iterations is               %d\n",*iterations);
  printf("Hidden Layer Size is                  %d\n",*sizeHiddenLayer);


} /* end readCommandLineArgs */


void readMatrixFromFile( char *fileName, 
                         float *matrix, 
                         int const rows, 
                         int const cols,
                         int const ld )
{
  FILE *ifp;

  ifp = fopen( fileName, "r" );

  if( ifp == NULL ) 
  {
    fprintf(stderr, "Error opening file %s\n", fileName);
    exit(911);
  } /* end if */

  for( int col = 0; col < cols; col++ )
  {
    for( int row = 0; row < rows; row++ )
    {
      if( !fscanf( ifp, "%f", 
          &matrix[ INDX( row, col, ld ) ] ) )
      {
        fprintf(stderr,"error reading training matrix file \n");
        exit(911);
      } /* end if */
    } /* end for row */
  } /* end for col */

  fclose(ifp);
  return;
} /* end readMatrixFromFile */
